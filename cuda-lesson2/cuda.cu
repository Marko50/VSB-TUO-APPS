#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "pic_type.h"


__global__ void kerner_rotate(CudaPic ori, CudaPic dest, float2 angle_props){
	// X,Y coordinates and check image dimensions
	int l_rotated_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_rotated_x = blockDim.x * blockIdx.x + threadIdx.y;
	
	if ( l_rotated_y >= dest.m_size.y ) return;
	if ( l_rotated_x >= dest.m_size.x ) return;
	
	float t_sin = angle_props.x;
	float t_cos = angle_props.y;

	int l_crotate_x = l_rotated_x - dest.m_size.x / 2;
	int l_crotate_y = l_rotated_y - dest.m_size.y / 2;

	// position in orig image
	float l_corig_x = t_cos * l_crotate_x - t_sin * l_crotate_y;
	float l_corig_y = t_sin * l_crotate_x + t_cos * l_crotate_y;
	// recalculation from centerpoint coordinates to image coordinates
	int l_orig_x = l_corig_x + ori.m_size.x / 2;
	int l_orig_y = l_corig_y + ori.m_size.y / 2;
	// out of orig image?
	if ( l_orig_y >= ori.m_size.y || l_orig_y < 0) return;
	if ( l_orig_x >= ori.m_size.x || l_orig_x < 0 ) return;

	dest.at<uchar3>( l_rotated_y, l_rotated_x ) = ori.at<uchar3>( l_orig_y, l_orig_x );
}

void cu_rotate(CudaPic ori, CudaPic dest, float2 angle_props, int t_square_size ){
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	dim3 l_blocks( ( dest.m_size.x + t_square_size - 1 ) / t_square_size,
			       ( dest.m_size.y + t_square_size - 1 ) / t_square_size );
	dim3 l_threads( t_square_size, t_square_size );
	kerner_rotate<<< l_blocks, l_threads >>>( ori, dest, angle_props );
	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

__global__ void kernel_bilin_scale(CudaPic ori, CudaPic dest){
	// X,Y coordinates and check image dimensions
	int l_resize_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_resize_x = blockDim.x * blockIdx.x + threadIdx.x;
	
	if ( l_resize_y >= dest.m_size.y ) return;
	if ( l_resize_x >= dest.m_size.x ) return;

	float l_scale_x = ori.m_size.x - 1;
	float l_scale_y = ori.m_size.y - 1;
	
	l_scale_x /= dest.m_size.x;
	l_scale_y /= dest.m_size.y;

	// new real position
	float l_orig_x = l_resize_x * l_scale_x;
	float l_orig_y = l_resize_y * l_scale_y;

	if ( l_orig_y >= ori.m_size.y || l_orig_y < 0) return;
	if ( l_orig_x >= ori.m_size.x || l_orig_x < 0 ) return;

	// diff x and y
	float l_diff_x = l_orig_x - ( int ) l_orig_x;
	float l_diff_y = l_orig_y - ( int ) l_orig_y;
	
	// points
	uchar3 bgr00 = ori.at<uchar3>( ( int ) l_orig_y, ( int ) l_orig_x );
	uchar3 bgr01 = ori.at<uchar3>( ( int ) l_orig_y, 1 + ( int ) l_orig_x );
	uchar3 bgr10 = ori.at<uchar3>( 1 + ( int ) l_orig_y, ( int ) l_orig_x );
	uchar3 bgr11 = ori.at<uchar3>( 1 + ( int ) l_orig_y, 1 + ( int ) l_orig_x );
	
	uchar3 bgr;
	bgr.x = bgr00.x * ( 1 - l_diff_y ) * ( 1 - l_diff_x ) + bgr01.x * ( 1 - l_diff_y ) * ( l_diff_x ) + bgr10.x * ( l_diff_y ) * ( 1 - l_diff_x ) + bgr11.x * ( l_diff_y ) * ( l_diff_x );
	bgr.y = bgr00.y * ( 1 - l_diff_y ) * ( 1 - l_diff_x ) + bgr01.y * ( 1 - l_diff_y ) * ( l_diff_x ) + bgr10.y * ( l_diff_y ) * ( 1 - l_diff_x ) + bgr11.y * ( l_diff_y ) * ( l_diff_x );
	bgr.z = bgr00.z * ( 1 - l_diff_y ) * ( 1 - l_diff_x ) + bgr01.z * ( 1 - l_diff_y ) * ( l_diff_x ) + bgr10.z * ( l_diff_y ) * ( 1 - l_diff_x ) + bgr11.z * ( l_diff_y ) * ( l_diff_x );

	dest.at<uchar3>(l_resize_y ,  l_resize_x) = bgr;
}

void cu_bilin_scale( CudaPic ori, CudaPic dest,  int t_square_size )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	dim3 l_blocks( ( dest.m_size.x + t_square_size - 1 ) / t_square_size,
			       ( dest.m_size.y + t_square_size - 1 ) / t_square_size );
	dim3 l_threads( t_square_size, t_square_size );
	kernel_bilin_scale<<< l_blocks, l_threads >>>( ori, dest );
	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// Demo kernel to create chess board
__global__ void kernel_chessboard( CudaPic t_color_pic )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	unsigned char b_or_w = 255 * ( ( blockIdx.x + blockIdx.y ) & 1 );

	// Store point into image
	t_color_pic.at<uchar3>(l_y, l_x) =  { b_or_w, b_or_w, b_or_w };
}

void cu_create_chessboard( CudaPic t_color_pic, int t_square_size )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	dim3 l_blocks( ( t_color_pic.m_size.x + t_square_size - 1 ) / t_square_size,
			       ( t_color_pic.m_size.y + t_square_size - 1 ) / t_square_size );
	dim3 l_threads( t_square_size, t_square_size );
	kernel_chessboard<<< l_blocks, l_threads >>>( t_color_pic );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

//Demo kernel to create picture with alpha channel gradient
__global__ void kernel_alphaimg( CudaPic t_color_pic, uchar3 t_color )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_pic.m_size.y ) return;
	if ( l_x >= t_color_pic.m_size.x ) return;

	int l_diagonal = sqrtf( t_color_pic.m_size.x * t_color_pic.m_size.x + t_color_pic.m_size.y * t_color_pic.m_size.y );
	int l_dx = l_x - t_color_pic.m_size.x / 2;
	int l_dy = l_y - t_color_pic.m_size.y / 2;
	int l_dxy = sqrtf( l_dx * l_dx + l_dy * l_dy ) - l_diagonal / 2;

	// Store point into image
	t_color_pic.at<uchar4>( l_y ,l_x ) =
		{ t_color.x, t_color.y, t_color.z, ( unsigned char ) ( 255 - 255 * l_dxy / ( l_diagonal / 2 ) ) };
}

void cu_create_alphaimg( CudaPic t_color_pic, uchar3 t_color )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_color_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_color_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_alphaimg<<< l_blocks, l_threads >>>( t_color_pic, t_color );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	// X,Y coordinates and check image dimensions
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_small_pic.m_size.y ) return;
	if ( l_x >= t_small_pic.m_size.x ) return;
	int l_by = l_y + t_position.y;
	int l_bx = l_x + t_position.x;
	if ( l_by >= t_big_pic.m_size.y || l_by < 0 ) return;
	if ( l_bx >= t_big_pic.m_size.x || l_bx < 0 ) return;

	// Get point from small image
	uchar4 l_fg_bgra = t_small_pic.at<uchar4>( l_y , l_x );
	uchar3 l_bg_bgr = t_big_pic.at<uchar3>( l_by , l_bx );
	uchar3 l_bgr = { 0, 0, 0 };

	// compose point from small and big image according alpha channel
	l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

	// Store point into image
	t_big_pic.at<uchar3>( l_by ,l_bx ) = l_bgr;
}

void cu_insertimage( CudaPic t_big_pic, CudaPic t_small_pic, int2 t_position )
{
	hipError_t l_cerr;

	// Grid creation, size of grid must be equal or greater than images
	int l_block_size = 32;
	dim3 l_blocks( ( t_small_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( t_big_pic, t_small_pic, t_position );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}
